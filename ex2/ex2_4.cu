
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1000
#define Nth 32

void random_ints(int* a, int size){
  for(int i =0; i<size; i++)
    a[i]=rand()%1000;
}

__global__ void addVecs(int *c, int *a, int *b, int L){
  int index = threadIdx.x + blockIdx.x *  blockDim.x;
  if(index < L)
    c[index] = a[index]+b[index];
}

int main(void){
  int *a, *b, *c;                    // host pointers
  int *d_a, *d_b, *d_c;        // device pointers
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c  
  hipMalloc((void **)&d_a, size); 
  hipMalloc((void **)&d_b, size); 
  hipMalloc((void **)&d_c, size); 

  a = (int *)malloc(size);
  random_ints(a, N);   // Alloc space host, random initialization
  b = (int *)malloc(size);
  random_ints(b, N);
  c = (int *)malloc(size); 

  // Copy data from host to device memory
  // cudaMemcpyHostToDevice is a flag determining copying from host to dev.
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch kernel to add two vector with 1 thread and N blocks
  // Kernel calls are asynchronous
  addVecs<<<(N+Nth-1)/Nth,Nth>>>(d_c, d_a, d_b, N);

  // Copy results from device to host
  // cudaMemcpy blocks CPU until Kernels finish execution
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i =0; i<N; i++)
    printf("%d + %d = %d\n",a[i],b[i],c[i]);

  
  // needs cudaFree to deallocate device pointers
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  free(a); free(b); free(c);
  return 0;
}
