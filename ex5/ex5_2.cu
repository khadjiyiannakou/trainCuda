
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line )
{
   hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Cuda Kernels are async. and one should use cudaDeviceSynchronize 
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    return;
}

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
   if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
   return;
}

#define N 512

void random_ints(int* a, int size){
  for(int i =0; i<size; i++)
    a[i]=rand()%1000;
}

__global__ void addVecs(int *c, int *a, int *b){
  c[threadIdx.x+N*N*N] = a[threadIdx.x]+b[threadIdx.x]; //access memory beyond boundaries
}


int main(){
  int *a, *b, *c;                    // host pointers
  int *d_a, *d_b, *d_c;        // device pointers
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c  
  hipMalloc((void **)&d_a, size); 
  hipMalloc((void **)&d_b, size); 
  hipMalloc((void **)&d_c, size); 

  a = (int *)malloc(size);
  random_ints(a, N);   // Alloc space host, random initialization
  b = (int *)malloc(size);
  random_ints(b, N);
  c = (int *)malloc(size); 

  // Copy data from host to device memory
  // cudaMemcpyHostToDevice is a flag determining copying from host to dev.
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch kernel to add two vector with 1 thread and N blocks
  // Kernel calls are asynchronous
  addVecs<<<1,N>>>(d_c, d_a, d_b);
  CudaCheckError();

  // Copy results from device to host
  // cudaMemcpy blocks CPU until Kernels finish execution
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i =0; i<N; i++)
    printf("%d + %d = %d\n",a[i],b[i],c[i]);

  
  // needs cudaFree to deallocate device pointers
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  free(a); free(b); free(c);
  return 0;


}
