
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (100*1024*1024)
#define Nth 1024

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
   if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
   return;
}

void random_ints(int* a, int size){
  for(int i =0; i<size; i++)
    a[i]=rand()%1000;
}

__global__ void addVecs(int *c, int *a, int *b, int L){
  int index = threadIdx.x + blockIdx.x *  blockDim.x;
  if(index < L)
    c[index] = a[index]+b[index];
}

int main(void){
  int *a=nullptr, *b=nullptr, *c=nullptr;                    // host pointers
  int *d_a, *d_b, *d_c;        // device pointers
  size_t size = N * sizeof(int);

  CudaSafeCall(hipSetDeviceFlags(hipDeviceMapHost));
  
  CudaSafeCall(hipHostAlloc((void**) &a, size,hipHostMallocMapped));// cudaHostAllocMapped));
  CudaSafeCall(hipHostAlloc((void**) &b, size, hipHostMallocMapped));
  CudaSafeCall(hipHostAlloc((void**) &c, size, hipHostMallocMapped));

  random_ints(a, N);
  random_ints(b, N);

  
  hipHostGetDevicePointer(&d_a,a,0);
  hipHostGetDevicePointer(&d_b,b,0);
  hipHostGetDevicePointer(&d_c,c,0);
  

  hipEvent_t start, stop;
  hipEventCreate(&start);hipEventCreate(&stop);
  hipEventRecord(start,0);
  
  // Launch kernel to add two vector with 1 thread and N blocks
  // Kernel calls are asynchronous
  addVecs<<<(N+Nth-1)/Nth,Nth>>>(d_c, d_a, d_b, N);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,start,stop);
  printf("Elapsed time is %f ms\n",elapsedTime);
  printf("Last element is %d\n",c[N-1]);
  hipEventDestroy(start);hipEventDestroy(stop);
  
  // needs cudaFree to deallocate host pointers which are allocated with cudaHostAlloc
  hipFree(a); hipFree(b); hipFree(c);
  return 0;
}
