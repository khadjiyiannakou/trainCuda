
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (100*1024*1024)
#define Nth 1024

void random_ints(int* a, int size){
  for(int i =0; i<size; i++)
    a[i]=rand()%1000;
}

__global__ void addVecs(int *c, int *a, int *b, int L){
  int index = threadIdx.x + blockIdx.x *  blockDim.x;
  if(index < L)
    c[index] = a[index]+b[index];
}

int main(void){
  int *a, *b, *c;                    // host pointers
  int *d_a, *d_b, *d_c;        // device pointers
  int size = N * sizeof(int);


  a = (int *)malloc(size);
  random_ints(a, N);   // Alloc space host, random initialization
  b = (int *)malloc(size);
  random_ints(b, N);
  c = (int *)malloc(size); 

  hipEvent_t start, stop;
  hipEventCreate(&start);hipEventCreate(&stop);
  hipEventRecord(start,0);
  // Alloc space for device copies of a, b, c  
  hipMalloc((void **)&d_a, size); 
  hipMalloc((void **)&d_b, size); 
  hipMalloc((void **)&d_c, size); 
  
  // Copy data from host to device memory
  // cudaMemcpyHostToDevice is a flag determining copying from host to dev.
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch kernel to add two vector with 1 thread and N blocks
  // Kernel calls are asynchronous
  addVecs<<<(N+Nth-1)/Nth,Nth>>>(d_c, d_a, d_b, N);

  // Copy results from device to host
  // cudaMemcpy blocks CPU until Kernels finish execution
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // for(int i =0; i<N; i++)
  //   printf("%d + %d = %d\n",a[i],b[i],c[i]);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,start,stop);
  printf("Elapsed time is %f ms\n",elapsedTime);
  printf("Last element is %d\n",c[N-1]);
  hipEventDestroy(start);hipEventDestroy(stop);
  
  // needs cudaFree to deallocate device pointers
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  free(a); free(b); free(c);
  return 0;
}
