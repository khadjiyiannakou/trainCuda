
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (100*1024*1024)
#define CHUNK_SIZE (1024*1024)

void random_ints(int* a, int size){
  for(int i =0; i<size; i++)
    a[i]=rand()%1000;
}

__global__ void addVecs(int *c, int *a, int *b){
  int index = threadIdx.x + blockIdx.x *  blockDim.x;
  c[index] = a[index]+b[index];
}

int main(){
  int *h_x, *h_y, *h_z;
  int *d_x0, *d_y0, *d_z0; // for stream 0
  int *d_x1, *d_y1, *d_z1; // for stream 1

  // Allocate page-locked host memory
  hipHostAlloc((void**)&h_x, N*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void**)&h_y, N*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void**)&h_z, N*sizeof(int), hipHostMallocDefault);

  // initialize vectors with random numbers
  random_ints(h_x,N);
  random_ints(h_y,N);

  
  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);

  // Allocate device memory
  hipMalloc((void**)&d_x0, CHUNK_SIZE*sizeof(int));
  hipMalloc((void**)&d_y0, CHUNK_SIZE*sizeof(int));
  hipMalloc((void**)&d_z0, CHUNK_SIZE*sizeof(int));

  hipMalloc((void**)&d_x1, CHUNK_SIZE*sizeof(int));
  hipMalloc((void**)&d_y1, CHUNK_SIZE*sizeof(int));
  hipMalloc((void**)&d_z1, CHUNK_SIZE*sizeof(int));



  for(int i = 0; i < N ; i += 2* CHUNK_SIZE){
    // operations on stream0
    hipMemcpyAsync(d_x0, h_x+i, CHUNK_SIZE*sizeof(int), hipMemcpyHostToDevice,stream0);
    hipMemcpyAsync(d_y0, h_y+i, CHUNK_SIZE*sizeof(int), hipMemcpyHostToDevice,stream0);
    addVecs<<<CHUNK_SIZE/1024,CHUNK_SIZE/1024, 0, stream0>>>(d_z0, d_x0, d_y0);
    hipMemcpyAsync(h_z+i, d_z0, CHUNK_SIZE*sizeof(int), hipMemcpyDeviceToHost,stream0);

    // operations on stream1
    hipMemcpyAsync(d_x1, h_x+i+CHUNK_SIZE, CHUNK_SIZE*sizeof(int), hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(d_y1, h_y+i+CHUNK_SIZE, CHUNK_SIZE*sizeof(int), hipMemcpyHostToDevice,stream1);
    addVecs<<<CHUNK_SIZE/1024,CHUNK_SIZE/1024, 0, stream1>>>(d_z1, d_x1, d_y1);
    hipMemcpyAsync(h_z+i+CHUNK_SIZE, d_z1, CHUNK_SIZE*sizeof(int), hipMemcpyDeviceToHost,stream1);
  }
  // we need to sync both streams
  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Elapsed Time is %f ms \n",elapsedTime);
  printf("Last element is %d\n",h_z[N-1]);
  hipHostFree(h_x); hipHostFree(h_y); hipHostFree(h_z);
  hipFree(d_x0);  hipFree(d_y0); hipFree(d_z0); hipFree(d_x1);  hipFree(d_y1); hipFree(d_z1);
  hipStreamDestroy(stream0); hipStreamDestroy(stream1); hipEventDestroy(start); hipEventDestroy(stop);
  return 0;
}
